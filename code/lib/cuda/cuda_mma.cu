#include "hip/hip_runtime.h"
#include "stdafx.h"
#define KERNEL_UNIT "cuda_mma/"
#include "cuda_mma.cuh"
#include "cuda_graph.cuh"
#include <lib/random/rand_utils.h>
#include <lib/math/matrix_utils.h>


namespace NCuda
{
__global__ void TestIntMMA(TCuda2DPtr<i8> a, TCuda2DPtr<i8> b, TCuda2DPtr<int> ab)
{
    //__shared__ T8SMemI8Tile shA;
    //__shared__ T8SMemI8Tile shB;
    //Copy8Tile(&shA, a);
    //Copy8Tile(&shB, b);
    __shared__ T4SMemI8Tile shA;
    __shared__ T4SMemI8Tile shB;
    Copy4Tile(&shA, a);
    Copy4Tile(&shB, b);

    TTileCoord tc;
    TRegTile<int> res;
    res.Clear();
    TRegTile<i8> tileA;
    TRegTile<i8> tileB;
    for (int k = 0; k < 4; ++k) {
        LoadTile(&tileA, shA, k);
        LoadTile(&tileB, shB, k);
        MMA(&res, tileA, tileB);
    }
    res.Store(tc, ab);
}



template <class TRng, class T>
static void InitRandomMatrix(TRng &rng, TArray2D<T> *pRes, yint xSize, yint ySize)
{
    pRes->SetSizes(xSize, ySize);
    for (yint y = 0; y < ySize; ++y) {
        for (yint x = 0; x < xSize; ++x) {
            (*pRes)[y][x] = rng.Uniform(100) - 50;
        }
    }
}


template <class T>
static TArray2D<double> Convert(const TArray2D<T> &matr)
{
    yint xSize = matr.GetXSize();
    yint ySize = matr.GetYSize();
    TArray2D<double> res;
    res.SetSizes(xSize, ySize);
    for (yint y = 0; y < ySize; ++y) {
        for (yint x = 0; x < xSize; ++x) {
            res[y][x] = matr[y][x];
        }
    }
    return res;
}

}
using namespace NCuda;
void TestMMA()
{
    TStream stream;

    TMersenne<ui32> rng(1313);
    TIntrusivePtr<TGraph> c = new TGraph;
    TCuda2DArray<i8> a;
    TCuda2DArray<i8> b;
    TCuda2DArray<int> ab;
    a.Allocate(64, 16);
    b.Allocate(64, 16);
    ab.Allocate(16, 16);

    CudaCall(c, TestIntMMA)(a, b).Write(&ab);

    for (;;) {
        TArray2D<i8> refA;
        TArray2D<i8> refB;
        InitRandomMatrix(rng, &refA, a.GetXSize(), a.GetYSize());
        InitRandomMatrix(rng, &refB, b.GetXSize(), b.GetYSize());
        a.Put(stream, refA);
        b.Put(stream, refB);

        c->Run(stream);
        ab.CopyToHost(stream);
        stream.Sync();

        TArray2D<double> refAB;
        MatrixMult(Convert(refA), Transpose(Convert(refB)), &refAB);
        TArray2D<int> gpuAB;
        ab.GetAllData(&gpuAB);
        for (yint y = 0; y < gpuAB.GetYSize(); ++y) {
            for (yint x = 0; x < gpuAB.GetXSize(); ++x) {
                Y_VERIFY(refAB[y][x] == gpuAB[y][x]);
            }
        }
        printf(".");
    }
}
